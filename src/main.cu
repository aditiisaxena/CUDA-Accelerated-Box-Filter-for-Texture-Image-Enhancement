#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "image_utils.h"

__global__ void sharpenKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * width + x;
    if (x > 0 && y > 0 && x < width-1 && y < height-1) {
        int val =
            -input[(y-1)*width + (x-1)] - input[(y-1)*width + x] - input[(y-1)*width + (x+1)] +
            -input[y*width + (x-1)] + 9*input[y*width + x] - input[y*width + (x+1)] +
            -input[(y+1)*width + (x-1)] - input[(y+1)*width + x] - input[(y+1)*width + (x+1)];

        output[idx] = min(max(val, 0), 255);
    } else if (x < width && y < height) {
        output[idx] = input[idx];
    }
}

void processImage(const cv::Mat& img, cv::Mat& out, hipStream_t stream) {
    int imgSize = img.rows * img.cols;
    unsigned char *d_in, *d_out;

    hipMallocAsync(&d_in, imgSize, stream);
    hipMallocAsync(&d_out, imgSize, stream);

    hipMemcpyAsync(d_in, img.data, imgSize, hipMemcpyHostToDevice, stream);

    dim3 threads(16, 16);
    dim3 blocks((img.cols + 15) / 16, (img.rows + 15) / 16);
    sharpenKernel<<<blocks, threads, 0, stream>>>(d_in, d_out, img.cols, img.rows);

    hipMemcpyAsync(out.data, d_out, imgSize, hipMemcpyDeviceToHost, stream);

    hipFreeAsync(d_in, stream);
    hipFreeAsync(d_out, stream);
}

int main() {
    std::string inputDir = "images/";
    std::string outputDir = "output/";

    auto files = getTiffFiles(inputDir);
    const int streamCount = 4;
    hipStream_t streams[streamCount];

    for (int i = 0; i < streamCount; ++i)
        hipStreamCreate(&streams[i]);

    for (size_t i = 0; i < files.size(); ++i) {
        cv::Mat img = readImageCleaned(files[i]);
        if (img.empty()) continue;

        cv::Mat result = img.clone();
        int streamIdx = i % streamCount;

        processImage(img, result, streams[streamIdx]);

        std::string filename = files[i].substr(files[i].find_last_of("/\\") + 1);
        std::string outPath = outputDir + filename;
        saveImage(outPath, result);
    }

    hipDeviceSynchronize();
    for (int i = 0; i < streamCount; ++i)
        hipStreamDestroy(streams[i]);

    std::cout << "All images processed.\n";
    return 0;
}